#include "hip/hip_runtime.h"
// http://www.caam.rice.edu/~timwar/CAAM210/Flows.html

#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#include "hip/hip_runtime.h"


extern "C"
{
#include "png_util.h"
}

#define dfloat float
// float

#define FLUID 0
#define WALL 1

#define NSPECIES 9

#ifndef NHALO 
#define NHALO 4
#define NSUBSTEPS (NHALO)
#endif

#ifndef TX
#define TX 32
#define TY 29
#endif

#include "hip/hip_runtime.h"

// loop up 1D array index from 2D node coordinates
__host__ __device__ __inline__ int idx(int N, int n, int m){
  return n + m*(N+2);
}

void lbmInput(const char *imageFileName,
	      dfloat threshold,
	      int *outN,
	      int *outM,
	      unsigned char **rgb,
	      unsigned char **alpha,
	      int **nodeType){

  int n,m, N,M;

  // read png file
  read_png(imageFileName, &N, &M, rgb, alpha);

  // pad to guarantee space around obstacle and extend the wake
  int Npad = 3*N;
  int Mpad = 2*M;

  if(Npad>8192) Npad = 8192;
  if(Mpad>8192) Mpad = 8192;

  // threshold walls based on gray scale
  *nodeType = (int*) calloc((Npad+2)*(Mpad+2), sizeof(int));

  // mark pixels by gray scale intensity
  unsigned char *rgbPad   = (unsigned char*) calloc(3*(Npad+2)*(Mpad+2), sizeof(unsigned char));
  unsigned char *alphaPad = (unsigned char*) calloc((Npad+2)*(Mpad+2),   sizeof(unsigned char));
  int wallCount = 0;
  for(m=1;m<=M;++m){
    for(n=1;n<=N;++n){
      int offset = ((n-1)+(m-1)*N);
      dfloat r = (*rgb)[3*offset+0];
      dfloat g = (*rgb)[3*offset+1];
      dfloat b = (*rgb)[3*offset+2];
      dfloat a = (*alpha) ? (*alpha)[offset]:255;
      // center image in padded region (including halo zone)

      int hoffset = N/4, yoffset = M/2;

      int id = idx(Npad,n+hoffset,m+yoffset);

      if(a==0)
	(*nodeType)[id] = FLUID;
      else
	(*nodeType)[id] = WALL*(sqrt(r*r+g*g+b*b)<threshold);

      wallCount += (*nodeType)[id];
      rgbPad[3*id+0] = r;
      rgbPad[3*id+1] = g;
      rgbPad[3*id+2] = b;
      alphaPad[id] = 255;
    }
  }

  for(n=1;n<=Npad;++n){
    (*nodeType)[idx(Npad,n,1)] = WALL;
    (*nodeType)[idx(Npad,n,Mpad)] = WALL;
  }
  
  free(*rgb); free(*alpha);
  *rgb = rgbPad;
  *alpha = alphaPad;
  
//  printf("wallCount = %d (%g percent of %d x %d nodes)\n", wallCount, 100.*((dfloat)wallCount/((Npad+2)*(Mpad+2))), Npad, Mpad);
  
  *outN = Npad;
  *outM = Mpad;
}

  
void lbmOutput(const char *fname,
	       const int *nodeType,
	       unsigned char *rgb,
	       unsigned char *alpha,
	       const dfloat c,
	       const dfloat dx,
	       int N,
	       int M,
	       const dfloat *f){
  int n,m,s;
  FILE *bah = fopen(fname, "w");

  // compute vorticity
  dfloat *Ux = (dfloat*) calloc((N+2)*(M+2), sizeof(dfloat));
  dfloat *Uy = (dfloat*) calloc((N+2)*(M+2), sizeof(dfloat));

  dfloat fnm[NSPECIES];
  for(m=1;m<=M;++m){
    for(n=1;n<=N;++n){
      int base = idx(N, n, m);
      for(s=0;s<NSPECIES;++s)
	fnm[s] = f[base+s*(N+2)*(M+2)];
      
      const dfloat rho = fnm[0]+fnm[1]+fnm[2]+fnm[3]+fnm[4]+fnm[5]+fnm[6]+fnm[7]+fnm[8];
      // macroscopic momentum
      Ux[base] = (fnm[1] - fnm[3] + fnm[5] - fnm[6] - fnm[7] + fnm[8])*c/rho;
      Uy[base] = (fnm[2] - fnm[4] + fnm[5] + fnm[6] - fnm[7] - fnm[8])*c/rho;
    }
  }

  
  dfloat plotMin = -4, plotMax = 4;
  for(m=1;m<=M;++m){
    for(n=1;n<=N;++n){
      int id = idx(N,n,m);

      // over write pixels in fluid region
      if(nodeType[id]==FLUID){
	unsigned char r,g,b,a;

	// reconstruct macroscopic density
	dfloat rho = 0;
	for(s=0;s<NSPECIES;++s)
	  rho += f[id+s*(N+2)*(M+2)];

	rho = ((rho-plotMin)/(plotMax-plotMin)); // rescale

	dfloat dUxdy = (Ux[idx(N,n,m+1)]-Ux[idx(N,n,m-1)])/(2.*dx);
	dfloat dUydx = (Uy[idx(N,n+1,m)]-Uy[idx(N,n-1,m)])/(2.*dx);
	
	dfloat curlU = dUydx-dUxdy;
	curlU = ((curlU-plotMin)/(plotMax-plotMin));

#if 0
	r = 255*curlU;
	g = 255*curlU;
	b = 255*curlU;
	a = 255;
#else
	a = 255;
	if(curlU>.55){
	  r = 255*(curlU-.55)/.45;
	  g = 0;
	  b = 0;
	}
	else if(curlU<.45){
	  r = 0;
	  g = 0;
	  b = 255*(.45-curlU)/.45;
	}
	else{
	  r = 255;
	  g = 255;
	  b = 255;
	}
#endif
	rgb[idx(N,n,m)*3+0] = r;
	rgb[idx(N,n,m)*3+1] = g;
	rgb[idx(N,n,m)*3+2] = b;
	alpha[idx(N,n,m)] = a;
      }
    }
  }
  
  write_png(bah, N+2, M+2, rgb, alpha);

  fclose(bah);
  free(Ux);
  free(Uy);
}


// weights used to compute equilibrium distribution (post collision)

#if 0
const dfloat w0 = 4.f/9.f, w1 = 1.f/9.f, w2 = 1.f/9.f, w3 =  1.f/9.f;
const dfloat w4 = 1.f/9.f, w5 = 1.f/36.f, w6 = 1.f/36.f, w7 = 1.f/36.f, w8 = 1.f/36.f;
#else
#define w0 (4.f/9.f)
#define w1 (1.f/9.f)
#define w2 (1.f/9.f)
#define w3 (1.f/9.f)
#define w4 (1.f/9.f)
#define w5 (1.f/36.f)
#define w6 (1.f/36.f)
#define w7 (1.f/36.f)
#define w8 (1.f/36.f)
#endif

#if 0
const dfloat g0 = 1.f, g1 = -2.f, g2 = -2.f, g3 = -2.f, g4 = -2.f;
const dfloat g5 = 4.f, g6 = 4.f, g7 = 4.f, g8 = 4.f;
#else
#define g0 (1.f)
#define g1 (-2.f)
#define g2 (-2.f)
#define g3 (-2.f)
#define g4 (-2.f)
#define g5 (4.f)
#define g6 (4.f)
#define g7 (4.f)
#define g8 (4.f)
#endif

#define wg0 (4.f/9.f)
#define wg1 (-2.f/9.f)
#define wg2 (-2.f/9.f)
#define wg3 (-2.f/9.f)
#define wg4 (-2.f/9.f)
#define wg5 (1.f/9.f)
#define wg6 (1.f/9.f)
#define wg7 (1.f/9.f)
#define wg8 (1.f/9.f)



__host__ __device__ __inline__ void lbmEquilibrium(const dfloat invc,
						   const dfloat rho,
						   const dfloat Ux, 
						   const dfloat Uy,
						   dfloat *  feq){

  // resolve macroscopic velocity into lattice particle velocity directions
//  const dfloat v0 = 0;
  const dfloat v1 = +Ux*invc;
  const dfloat v2 = +Uy*invc;
  const dfloat v3 = -v1; // -Ux*invc;
  const dfloat v4 = -v2; // -Uy*invc;
  const dfloat v5 =  v1+v2;  // (+Ux+Uy)*invc;
  const dfloat v6 =  -v1+v2; // (-Ux+Uy)*invc;
  const dfloat v7 =  -v1-v2; // (-Ux-Uy)*invc;
  const dfloat v8 =   v1-v2; //  (+Ux-Uy)*invc;
  
  // compute LBM post-collisional
  const dfloat U2 = Ux*Ux+Uy*Uy;
  const dfloat shift = 1.f -1.5f*U2*invc*invc;
  feq[0] = rho*w0*(shift);
  feq[1] = rho*w1*(v1*(3.f + 4.5f*v1) + shift);
  feq[2] = rho*w2*(v2*(3.f + 4.5f*v2) + shift);
  feq[3] = rho*w3*(v3*(3.f + 4.5f*v3) + shift);
  feq[4] = rho*w4*(v4*(3.f + 4.5f*v4) + shift);
  feq[5] = rho*w5*(v5*(3.f + 4.5f*v5) + shift);
  feq[6] = rho*w6*(v6*(3.f + 4.5f*v6) + shift);
  feq[7] = rho*w7*(v7*(3.f + 4.5f*v7) + shift);
  feq[8] = rho*w8*(v8*(3.f + 4.5f*v8) + shift);
  
}


// perform lattice streaming and collision steps
__global__ void lbmUpdateV0(const int N,                  // number of nodes in x
			  const int M,                  // number of nodes in y
			  const dfloat c,                // speed of sound
			  const dfloat * __restrict__ tau,           // relaxation rate
			  const int    * __restrict__ nodeType,      // (N+2) x (M+2) node types 
			  const dfloat * __restrict__ f,             // (N+2) x (M+2) x 9 fields before streaming and collisions
			  dfloat * __restrict__ fnew){               // (N+2) x (M+2) x 9 fields after streaming and collisions
  
  // number of nodes in whole array including halo
  int Nall = (N+2)*(M+2);
  
  // loop over all non-halo nodes in lattice
  int n = 1 + threadIdx.x + blockIdx.x*TX;
  int m = 1 + threadIdx.y + blockIdx.y*TY;

  dfloat invc = 1.f/c;
  
  if(m<M+1 && n<=N+1){

    // physics paramaters
    dfloat tauinv = 1.f/tau[idx(N,n,m)];
    
    // discover type of node (WALL or FLUID)
    const int nt = nodeType[idx(N,n,m)];
    dfloat fnm[NSPECIES];
    
    // OUTFLOW
    if(n==N+1){
      fnm[0] = f[idx(N,n,  m)   + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n-1,m)   + 1*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m-1)   + 2*Nall]; // N bound from S
      fnm[3] = f[idx(N,n,m)     + 3*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m+1)   + 4*Nall]; // S bound from N
      fnm[5] = f[idx(N,n-1,m-1) + 5*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n,m-1)   + 6*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n,m+1)   + 7*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n-1,m+1) + 8*Nall]; // SE bound from NW      
    }
    else if(nt == FLUID){
      fnm[0] = f[idx(N,n,  m)   + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n-1,m)   + 1*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m-1)   + 2*Nall]; // N bound from S
      fnm[3] = f[idx(N,n+1,m)   + 3*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m+1)   + 4*Nall]; // S bound from N
      fnm[5] = f[idx(N,n-1,m-1) + 5*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n+1,m-1) + 6*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n+1,m+1) + 7*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n-1,m+1) + 8*Nall]; // SE bound from NW
    }
    else{
      // WALL reflects particles
      fnm[0] = f[idx(N,n,m) + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n,m) + 3*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m) + 4*Nall]; // N bound from S
      fnm[3] = f[idx(N,n,m) + 1*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m) + 2*Nall]; // S bound from N
      fnm[5] = f[idx(N,n,m) + 7*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n,m) + 8*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n,m) + 5*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n,m) + 6*Nall]; // SE bound from NW
    }
    
    // macroscopic density
    const dfloat rho = fnm[0]+fnm[1]+fnm[2]+fnm[3]+fnm[4]+fnm[5]+fnm[6]+fnm[7]+fnm[8];
    
    //    if(rho<1e-4){ printf("rho(%d,%d)=%g\n", n,m,rho); exit(-1); }
    
    // macroscopic momentum
    const dfloat delta2 = 1e-8;
    const dfloat Ux = (fnm[1] - fnm[3] + fnm[5] - fnm[6] - fnm[7] + fnm[8])*c/sqrt(rho*rho+delta2);
    const dfloat Uy = (fnm[2] - fnm[4] + fnm[5] + fnm[6] - fnm[7] - fnm[8])*c/sqrt(rho*rho+delta2);

    // compute equilibrium distribution
    dfloat feq[NSPECIES];
    lbmEquilibrium(invc, rho, Ux, Uy, feq);

    // MRT stabilization

    const dfloat R = g0*fnm[0] + g1*fnm[1] + g2*fnm[2]+ g3*fnm[3] + g4*fnm[4] + g5*fnm[5] + g6*fnm[6] + g7*fnm[7] + g8*fnm[8];
        
    // post collision densities
    dfloat fac = (1.f-tauinv)*R*0.25f;
    
    fnm[0] -= tauinv*(fnm[0]-feq[0]) + w0*g0*fac;
    fnm[1] -= tauinv*(fnm[1]-feq[1]) + w1*g1*fac;
    fnm[2] -= tauinv*(fnm[2]-feq[2]) + w2*g2*fac;
    fnm[3] -= tauinv*(fnm[3]-feq[3]) + w3*g3*fac;
    fnm[4] -= tauinv*(fnm[4]-feq[4]) + w4*g4*fac;
    fnm[5] -= tauinv*(fnm[5]-feq[5]) + w5*g5*fac;
    fnm[6] -= tauinv*(fnm[6]-feq[6]) + w6*g6*fac;
    fnm[7] -= tauinv*(fnm[7]-feq[7]) + w7*g7*fac;
    fnm[8] -= tauinv*(fnm[8]-feq[8]) + w8*g8*fac;
      
    // store new densities
    const int base = idx(N,n,m);
    fnew[base+0*Nall] = fnm[0];
    fnew[base+1*Nall] = fnm[1];
    fnew[base+2*Nall] = fnm[2];
    fnew[base+3*Nall] = fnm[3];
    fnew[base+4*Nall] = fnm[4];
    fnew[base+5*Nall] = fnm[5];
    fnew[base+6*Nall] = fnm[6];
    fnew[base+7*Nall] = fnm[7];
    fnew[base+8*Nall] = fnm[8];
  }
  
}



__global__ void lbmUpdateV1(const int N,                  // number of nodes in x
			    const int M,                  // number of nodes in y
			    const dfloat c,                // speed of sound
			    const dfloat * __restrict__ tau,           // relaxation rate
			    const int    * __restrict__ nodeType,      // (N+2) x (M+2) node types 
			    const dfloat * __restrict__ f,             // (N+2) x (M+2) x 9 fields before streaming and collisions
			    dfloat * __restrict__ fnew){               // (N+2) x (M+2) x 9 fields after streaming and collisions
  
  // number of nodes in whole array including halo
  int Nall = (N+2)*(M+2);
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  // loop over all non-halo nodes in lattice
  int n = -(NHALO) + 1 + tx + blockIdx.x*(TX-2*NHALO);
  int m = -(NHALO) + 1 + ty + blockIdx.y*(TY-2*NHALO);

  dfloat invc = 1.f/c;
  
  if(m>=1 && m<M+1 && n>=1 && n<=N+1){

    // physics paramaters
    dfloat tauinv = 1.f/tau[idx(N,n,m)];
    
    // discover type of node (WALL or FLUID)
    const int nt = nodeType[idx(N,n,m)];
    dfloat fnm[NSPECIES];
    
    // OUTFLOW
    if(n==N+1){
      fnm[0] = f[idx(N,n,  m)   + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n-1,m)   + 1*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m-1)   + 2*Nall]; // N bound from S
      fnm[3] = f[idx(N,n,m)     + 3*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m+1)   + 4*Nall]; // S bound from N
      fnm[5] = f[idx(N,n-1,m-1) + 5*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n,m-1)   + 6*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n,m+1)   + 7*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n-1,m+1) + 8*Nall]; // SE bound from NW      
    }
    else if(nt == FLUID){
      fnm[0] = f[idx(N,n,  m)   + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n-1,m)   + 1*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m-1)   + 2*Nall]; // N bound from S
      fnm[3] = f[idx(N,n+1,m)   + 3*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m+1)   + 4*Nall]; // S bound from N
      fnm[5] = f[idx(N,n-1,m-1) + 5*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n+1,m-1) + 6*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n+1,m+1) + 7*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n-1,m+1) + 8*Nall]; // SE bound from NW
    }
    else{
      // WALL reflects particles
      fnm[0] = f[idx(N,n,m) + 0*Nall]; // stationary 
      fnm[1] = f[idx(N,n,m) + 3*Nall]; // E bound from W
      fnm[2] = f[idx(N,n,m) + 4*Nall]; // N bound from S
      fnm[3] = f[idx(N,n,m) + 1*Nall]; // W bound from E
      fnm[4] = f[idx(N,n,m) + 2*Nall]; // S bound from N
      fnm[5] = f[idx(N,n,m) + 7*Nall]; // NE bound from SW
      fnm[6] = f[idx(N,n,m) + 8*Nall]; // NW bound from SE
      fnm[7] = f[idx(N,n,m) + 5*Nall]; // SW bound from NE
      fnm[8] = f[idx(N,n,m) + 6*Nall]; // SE bound from NW
    }
    
    // macroscopic density
    const dfloat rho = fnm[0]+fnm[1]+fnm[2]+fnm[3]+fnm[4]+fnm[5]+fnm[6]+fnm[7]+fnm[8];
    
    //    if(rho<1e-4){ printf("rho(%d,%d)=%g\n", n,m,rho); exit(-1); }
    
    // macroscopic momentum
    const dfloat delta2 = 1e-8;
    const dfloat Ux = (fnm[1] - fnm[3] + fnm[5] - fnm[6] - fnm[7] + fnm[8])*c/sqrt(rho*rho+delta2);
    const dfloat Uy = (fnm[2] - fnm[4] + fnm[5] + fnm[6] - fnm[7] - fnm[8])*c/sqrt(rho*rho+delta2);

    // compute equilibrium distribution
    dfloat feq[NSPECIES];
    lbmEquilibrium(invc, rho, Ux, Uy, feq);

    // MRT stabilization

    const dfloat R = g0*fnm[0] + g1*fnm[1] + g2*fnm[2]+ g3*fnm[3] + g4*fnm[4] + g5*fnm[5] + g6*fnm[6] + g7*fnm[7] + g8*fnm[8];
        
    // post collision densities
    fnm[0] -= tauinv*(fnm[0]-feq[0]) + (1.f-tauinv)*w0*g0*R*0.25f;
    fnm[1] -= tauinv*(fnm[1]-feq[1]) + (1.f-tauinv)*w1*g1*R*0.25f;
    fnm[2] -= tauinv*(fnm[2]-feq[2]) + (1.f-tauinv)*w2*g2*R*0.25f;
    fnm[3] -= tauinv*(fnm[3]-feq[3]) + (1.f-tauinv)*w3*g3*R*0.25f;
    fnm[4] -= tauinv*(fnm[4]-feq[4]) + (1.f-tauinv)*w4*g4*R*0.25f;
    fnm[5] -= tauinv*(fnm[5]-feq[5]) + (1.f-tauinv)*w5*g5*R*0.25f;
    fnm[6] -= tauinv*(fnm[6]-feq[6]) + (1.f-tauinv)*w6*g6*R*0.25f;
    fnm[7] -= tauinv*(fnm[7]-feq[7]) + (1.f-tauinv)*w7*g7*R*0.25f;
    fnm[8] -= tauinv*(fnm[8]-feq[8]) + (1.f-tauinv)*w8*g8*R*0.25f;
      
    // store new densities
    const int base = idx(N,n,m);
    if(tx>=NHALO && tx<TX-NHALO && ty>=NHALO && ty<TY-NHALO){
      fnew[base+0*Nall] = fnm[0];
      fnew[base+1*Nall] = fnm[1];
      fnew[base+2*Nall] = fnm[2];
      fnew[base+3*Nall] = fnm[3];
      fnew[base+4*Nall] = fnm[4];
      fnew[base+5*Nall] = fnm[5];
      fnew[base+6*Nall] = fnm[6];
      fnew[base+7*Nall] = fnm[7];
      fnew[base+8*Nall] = fnm[8];
    }
  }
  
}




__global__ void lbmUpdateV2(const int N,                  // number of nodes in x
			    const int M,                  // number of nodes in y
			    const dfloat c,                // speed of sound
			    const dfloat * __restrict__ tau,           // relaxation rate
			    const int    * __restrict__ nodeType,      // (N+2) x (M+2) node types 
			    const dfloat * __restrict__ f,             // (N+2) x (M+2) x 9 fields before streaming and collisions
			    dfloat * __restrict__ fnew){               // (N+2) x (M+2) x 9 fields after streaming and collisions

  __shared__ dfloat s_f[9][TY][TX];
  
  // number of nodes in whole array including halo
  int Nall = (N+2)*(M+2);
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  // loop over all non-halo nodes in lattice
  int n = -(NHALO) + 1 + tx + blockIdx.x*(TX-2*NHALO);
  int m = -(NHALO) + 1 + ty + blockIdx.y*(TY-2*NHALO);

  if(m>=0 && m<M+2 && n>=0 && n<=N+1){
    const int id = idx(N,n,m);
    for(int fld=0;fld<9;++fld){
      s_f[fld][ty][tx] = f[id + fld*Nall];     
    }
  }

  dfloat invc = 1.f/c;
  
  __syncthreads();
  
  if(m>=1 && m<M+1 && n>=1 && n<=N+1){
    if(tx>0 && tx<TX-1 && ty>0 && ty<TY-1){
      // physics paramaters
      dfloat tauinv = 1.f/tau[idx(N,n,m)];
      
      // discover type of node (WALL or FLUID)
      const int nt = nodeType[idx(N,n,m)];
      dfloat fnm[NSPECIES];
      
      // OUTFLOW
      if(n==N+1){
	fnm[0]  = s_f[0][ty][tx]     ; // stationary 
	fnm[1]  = s_f[1][ty][tx-1]   ; // E bound from W
	fnm[2]  = s_f[2][ty-1][tx]   ; // N bound from S
	fnm[3]  = s_f[3][ty][tx]     ; // W bound from E
	fnm[4]  = s_f[4][ty+1][tx]   ; // S bound from N
	fnm[5]  = s_f[5][ty-1][tx-1] ; // NE bound from SW
	fnm[6]  = s_f[6][ty-1][tx]   ; // NW bound from SE
	fnm[7]  = s_f[7][ty+1][tx]   ; // SW bound from NE
	fnm[8]  = s_f[8][ty+1][tx-1] ; // SE bound from NW      
      }
      else if(nt==FLUID){
	fnm[0]  = s_f[0][ty][tx]     ; // stationary 
	fnm[1]  = s_f[1][ty][tx-1]   ; // E bound from W
	fnm[2]  = s_f[2][ty-1][tx]   ; // N bound from S
	fnm[3]  = s_f[3][ty][tx+1]   ; // W bound from E
	fnm[4]  = s_f[4][ty+1][tx]   ; // S bound from N
	fnm[5]  = s_f[5][ty-1][tx-1] ; // NE bound from SW
	fnm[6]  = s_f[6][ty-1][tx+1] ; // NW bound from SE
	fnm[7]  = s_f[7][ty+1][tx+1] ; // SW bound from NE
	fnm[8]  = s_f[8][ty+1][tx-1] ; // SE bound from NW
      }
      else{
	// WALL reflects particles
	fnm[0]  = s_f[0][ty][tx]; // stationary 
	fnm[1]  = s_f[3][ty][tx]; // E bound from W
	fnm[2]  = s_f[4][ty][tx]; // N bound from S
	fnm[3]  = s_f[1][ty][tx]; // W bound from E
	fnm[4]  = s_f[2][ty][tx]; // S bound from N
	fnm[5]  = s_f[7][ty][tx]; // NE bound from SW
	fnm[6]  = s_f[8][ty][tx]; // NW bound from SE
	fnm[7]  = s_f[5][ty][tx]; // SW bound from NE
	fnm[8]  = s_f[6][ty][tx]; // SE bound from NW
      }
      
      // macroscopic density
      const dfloat rho = fnm[0]+fnm[1]+fnm[2]+fnm[3]+fnm[4]+fnm[5]+fnm[6]+fnm[7]+fnm[8];
      
      //    if(rho<1e-4){ printf("rho(%d,%d)=%g\n", n,m,rho); exit(-1); }
      
      // macroscopic momentum
      const dfloat delta2 = 1e-8;
      const dfloat Ux = (fnm[1] - fnm[3] + fnm[5] - fnm[6] - fnm[7] + fnm[8])*c/sqrt(rho*rho+delta2);
      const dfloat Uy = (fnm[2] - fnm[4] + fnm[5] + fnm[6] - fnm[7] - fnm[8])*c/sqrt(rho*rho+delta2);
      
      // compute equilibrium distribution
      dfloat feq[NSPECIES];
      lbmEquilibrium(invc, rho, Ux, Uy, feq);
      
      // MRT stabilization
      const dfloat R = g0*fnm[0] + g1*fnm[1] + g2*fnm[2]+ g3*fnm[3] + g4*fnm[4] + g5*fnm[5] + g6*fnm[6] + g7*fnm[7] + g8*fnm[8];
      
      // post collision densities
      fnm[0] -= tauinv*(fnm[0]-feq[0]) + (1.f-tauinv)*w0*g0*R*0.25f;
      fnm[1] -= tauinv*(fnm[1]-feq[1]) + (1.f-tauinv)*w1*g1*R*0.25f;
      fnm[2] -= tauinv*(fnm[2]-feq[2]) + (1.f-tauinv)*w2*g2*R*0.25f;
      fnm[3] -= tauinv*(fnm[3]-feq[3]) + (1.f-tauinv)*w3*g3*R*0.25f;
      fnm[4] -= tauinv*(fnm[4]-feq[4]) + (1.f-tauinv)*w4*g4*R*0.25f;
      fnm[5] -= tauinv*(fnm[5]-feq[5]) + (1.f-tauinv)*w5*g5*R*0.25f;
      fnm[6] -= tauinv*(fnm[6]-feq[6]) + (1.f-tauinv)*w6*g6*R*0.25f;
      fnm[7] -= tauinv*(fnm[7]-feq[7]) + (1.f-tauinv)*w7*g7*R*0.25f;
      fnm[8] -= tauinv*(fnm[8]-feq[8]) + (1.f-tauinv)*w8*g8*R*0.25f;
      
      // store new densities
      const int base = idx(N,n,m);
      if(tx>=NHALO && tx<TX-NHALO && ty>=NHALO && ty<TY-NHALO){
	fnew[base+0*Nall] = fnm[0];
	fnew[base+1*Nall] = fnm[1];
	fnew[base+2*Nall] = fnm[2];
	fnew[base+3*Nall] = fnm[3];
	fnew[base+4*Nall] = fnm[4];
	fnew[base+5*Nall] = fnm[5];
	fnew[base+6*Nall] = fnm[6];
	fnew[base+7*Nall] = fnm[7];
	fnew[base+8*Nall] = fnm[8];
      }
    }
  }
  
}

__global__ __launch_bounds__(TX*TY)
  void lbmUpdateV3(const int N,                  // number of nodes in x
		   const int M,                  // number of nodes in y
		   const dfloat c,                // speed of sound
		   const dfloat * __restrict__ tauInv,           // relaxation rate
		   const int    * __restrict__ nodeType,      // (N+2) x (M+2) node types 
		   const dfloat * __restrict__ f,             // (N+2) x (M+2) x 9 fields before streaming and collisions
		   dfloat * __restrict__ fnew){               // (N+2) x (M+2) x 9 fields after streaming and collisions
  
  __shared__ dfloat s_f[9][TY][TX+1];
  
  // number of nodes in whole array including halo
  const int Nall = (N+2)*(M+2);
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  
  // loop over all non-halo nodes in lattice
  const int n = -(NHALO) + 1 + tx + blockIdx.x*(TX-2*NHALO);
  const int m = -(NHALO) + 1 + ty + blockIdx.y*(TY-2*NHALO);

  const dfloat invc = 1.f/c;
  
  if(m>=0 && m<M+2 && n>=0 && n<=N+1){
    const int id = idx(N,n,m);

#pragma unroll
    for(int fld=0;fld<9;++fld){
      s_f[fld][ty][tx] = f[id + fld*Nall];     
    }
  }

  dfloat tauinv = 1;
  int nt = 0;
  
  if(m>=1 && m<M+1 && n>=1 && n<=N+1){
    if(tx>0 && tx<TX-1 && ty>0 && ty<TY-1){
      // physics paramaters
      tauinv = tauInv[idx(N,n,m)];
      
      // discover type of node (WALL or FLUID)
      nt = nodeType[idx(N,n,m)];
    }
  }

  dfloat fnm[NSPECIES];
  dfloat feq[NSPECIES];

  const int test = (m>=1 && m<M+1 && n>=1 && n<=N+1) && (tx>0 && tx<TX-1 && ty>0 && ty<TY-1);
  
#pragma unroll 
  for(int step=0;step<NSUBSTEPS;++step){
    
    __syncthreads(); 
    
    if(test){
      // OUTFLOW
      if(n==N+1){
	fnm[0]  = s_f[0][ty][tx]     ; // stationary 
	fnm[1]  = s_f[1][ty][tx-1]   ; // E bound from W
	fnm[2]  = s_f[2][ty-1][tx]   ; // N bound from S
	fnm[3]  = s_f[3][ty][tx]     ; // W bound from E
	fnm[4]  = s_f[4][ty+1][tx]   ; // S bound from N
	fnm[5]  = s_f[5][ty-1][tx-1] ; // NE bound from SW
	fnm[6]  = s_f[6][ty-1][tx]   ; // NW bound from SE
	fnm[7]  = s_f[7][ty+1][tx]   ; // SW bound from NE
	fnm[8]  = s_f[8][ty+1][tx-1] ; // SE bound from NW      
      }
      else if(nt==FLUID){
	fnm[0]  = s_f[0][ty][tx]     ; // stationary 
	fnm[1]  = s_f[1][ty][tx-1]   ; // E bound from W
	fnm[2]  = s_f[2][ty-1][tx]   ; // N bound from S
	fnm[3]  = s_f[3][ty][tx+1]   ; // W bound from E
	fnm[4]  = s_f[4][ty+1][tx]   ; // S bound from N
	fnm[5]  = s_f[5][ty-1][tx-1] ; // NE bound from SW
	fnm[6]  = s_f[6][ty-1][tx+1] ; // NW bound from SE
	fnm[7]  = s_f[7][ty+1][tx+1] ; // SW bound from NE
	fnm[8]  = s_f[8][ty+1][tx-1] ; // SE bound from NW
      }
      else{
	// WALL reflects particles
	fnm[0]  = s_f[0][ty][tx]; // stationary 
	fnm[1]  = s_f[3][ty][tx]; // E bound from W
	fnm[2]  = s_f[4][ty][tx]; // N bound from S
	fnm[3]  = s_f[1][ty][tx]; // W bound from E
	fnm[4]  = s_f[2][ty][tx]; // S bound from N
	fnm[5]  = s_f[7][ty][tx]; // NE bound from SW
	fnm[6]  = s_f[8][ty][tx]; // NW bound from SE
	fnm[7]  = s_f[5][ty][tx]; // SW bound from NE
	fnm[8]  = s_f[6][ty][tx]; // SE bound from NW
      }
	
      // macroscopic density
      const dfloat rho = fnm[0]+fnm[1]+fnm[2]+fnm[3]+fnm[4]+fnm[5]+fnm[6]+fnm[7]+fnm[8];

      const dfloat delta2 = 1e-8;
      const dfloat denom = c*rsqrt(rho*rho+delta2);
      //      const dfloat denom = c/(fabs(rho)+delta2);

      // macroscopic momentum
      const dfloat Ux = (fnm[1] - fnm[3] + fnm[5] - fnm[6] - fnm[7] + fnm[8])*denom;
      const dfloat Uy = (fnm[2] - fnm[4] + fnm[5] + fnm[6] - fnm[7] - fnm[8])*denom;
      
      // compute equilibrium distribution
      lbmEquilibrium(invc, rho, Ux, Uy, feq);
	
      // MRT stabilization
      dfloat R = g0*fnm[0];
      R += g1*fnm[1];
      R += g2*fnm[2];
      R += g3*fnm[3];
      R += g4*fnm[4];
      R += g5*fnm[5];
      R += g6*fnm[6];
      R += g7*fnm[7];
      R += g8*fnm[8];
	
      // post collision densities
      const dfloat fac = (1.f-tauinv)*R*0.25f;
      fnm[0] -= tauinv*(fnm[0]-feq[0]) + fac*w0*g0;
      fnm[1] -= tauinv*(fnm[1]-feq[1]) + fac*w1*g1;
      fnm[2] -= tauinv*(fnm[2]-feq[2]) + fac*w2*g2;
      fnm[3] -= tauinv*(fnm[3]-feq[3]) + fac*w3*g3;
      fnm[4] -= tauinv*(fnm[4]-feq[4]) + fac*w4*g4;
      fnm[5] -= tauinv*(fnm[5]-feq[5]) + fac*w5*g5;
      fnm[6] -= tauinv*(fnm[6]-feq[6]) + fac*w6*g6;
      fnm[7] -= tauinv*(fnm[7]-feq[7]) + fac*w7*g7;
      fnm[8] -= tauinv*(fnm[8]-feq[8]) + fac*w8*g8;
      
    }
    
    if(step<NSUBSTEPS-1){
      
      __syncthreads();
      
      if(test){
#pragma unroll
	for(int fld=0;fld<9;++fld){
	  s_f[fld][ty][tx] = fnm[fld];
	}
      }
    }
  }
  
  if(m>=1 && m<M+1 && n>=1 && n<=N+1){
    if(tx>=NHALO && tx<TX-NHALO && ty>=NHALO && ty<TY-NHALO){
      // store new densities
      const int base = idx(N,n,m);
      fnew[base+0*Nall] = fnm[0];
      fnew[base+1*Nall] = fnm[1];
      fnew[base+2*Nall] = fnm[2];
      fnew[base+3*Nall] = fnm[3];
      fnew[base+4*Nall] = fnm[4];
      fnew[base+5*Nall] = fnm[5];
      fnew[base+6*Nall] = fnm[6];
      fnew[base+7*Nall] = fnm[7];
      fnew[base+8*Nall] = fnm[8];
    }
  }
}




void lbmCheck(int N, int M, dfloat *f){

  int n,m,s;
  int nanCount = 0;
  for(s=0;s<NSPECIES;++s){
    for(m=0;m<=M+1;++m){
      for(n=0;n<=N+1;++n){

	nanCount += isnan(f[idx(N,n,m)+s*(N+2)*(M+2)]);
      }
    }
  }
  
  if(nanCount){   printf("found %d nans\n", nanCount); exit(-1); }
}



// set initial conditions (use uniform flow f everywhere)
void lbmInitialConditions(dfloat c, int N, int M, int *nodeType, dfloat *f){
  int n,m;
  dfloat feqIC[NSPECIES];
  dfloat feqWALL[NSPECIES];
  dfloat rhoIC = 1.;
  dfloat UxIC = 1.;
  dfloat UyIC = 0.;
  dfloat invc = 1./c;
  
  lbmEquilibrium(invc, rhoIC,    0.,  0., feqWALL);
  lbmEquilibrium(invc, rhoIC, UxIC, UyIC, feqIC);
  
  for(m=0;m<=M+1;++m){
    for(n=0;n<=N+1;++n){
      int base = idx(N, n, m);
      int s;

      if(n==0 || m==0 || m==M+1){
	for(s=0;s<NSPECIES;++s){
	  f[idx(N,n,m)+s*(N+2)*(M+2)] = feqIC[s];
	}
      }
      else{
	for(s=0;s<NSPECIES;++s){
	  f[idx(N,n,m)+s*(N+2)*(M+2)] = feqWALL[s];
	}
      }
    }
  }
}

int main(int argc, char **argv){

    hipSetDevice(1);

  if(argc!=3){
    printf("usage: ./lbm foo.png threshold\n");
    exit(-1);
  }

  // read threshold 
  dfloat threshold = atof(argv[2]);
  char *imageFileName = strdup(argv[1]);

  int N, M; // size of lattice
  unsigned char *rgb, *alpha;
  int *nodeType;
  lbmInput(imageFileName, threshold, &N, &M, &rgb, &alpha, &nodeType); 
  
  // physical parameters
  dfloat dx = .01;    // lattice node spacings 
  dfloat dt = dx*.1; // time step (also determines Mach number)
  dfloat c  = dx/dt; // speed of sound
  dfloat tau = .65; // relaxation rate
  dfloat Reynolds = 2./((tau-.5)*c*c*dt/3.);

//  printf("Reynolds number %g\n", Reynolds);

  // create lattice storage
  dfloat *h_f    = (dfloat*) calloc((N+2)*(M+2)*NSPECIES, sizeof(dfloat));
  dfloat *h_fnew = (dfloat*) calloc((N+2)*(M+2)*NSPECIES, sizeof(dfloat));
  dfloat *h_tau  = (dfloat*) calloc((N+2)*(M+2), sizeof(dfloat));
  dfloat *h_tauInv  = (dfloat*) calloc((N+2)*(M+2), sizeof(dfloat));
  
  // set initial flow densities
  lbmInitialConditions(c, N, M, nodeType, h_f);
  lbmInitialConditions(c, N, M, nodeType, h_fnew);

  // set tau based on n index
  dfloat xo = .95;
  int n,m;
  for(m=0;m<M+2;++m){
    for(n=0;n<N+2;++n){
      dfloat x = ((double)n)/N;
      dfloat taunm = tau*(1 + 4*(1+tanh(20*(x-xo))));
      h_tau[idx(N,n,m)] = taunm;
      h_tauInv[idx(N,n,m)] = 1.f/taunm;
    }
  }

  // DEVICE storage
  dfloat *c_f, *c_fnew, *c_tau,*c_tauInv;
  int *c_nodeType;
  
  hipMalloc(&c_f,    (N+2)*(M+2)*NSPECIES*sizeof(dfloat));
  hipMalloc(&c_fnew, (N+2)*(M+2)*NSPECIES*sizeof(dfloat));
  hipMalloc(&c_nodeType, (N+2)*(M+2)*sizeof(int));
  hipMalloc(&c_tau,      (N+2)*(M+2)*sizeof(dfloat));
  hipMalloc(&c_tauInv,      (N+2)*(M+2)*sizeof(dfloat));

  hipMemcpy(c_f,    h_f,    (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(c_fnew, h_fnew, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(c_nodeType, nodeType, (N+2)*(M+2)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(c_tau,  h_tau, (N+2)*(M+2)*sizeof(dfloat), hipMemcpyHostToDevice);
  hipMemcpy(c_tauInv,  h_tauInv, (N+2)*(M+2)*sizeof(dfloat), hipMemcpyHostToDevice);

  hipEvent_t tic, toc;
  hipEventCreate(&tic);
  hipEventCreate(&toc);

  // Use these settings for simulations
  int Nsteps = 480000/2, tstep = 0, iostep = 100;

  // I use these settings for tuning
  //  int Nsteps = 121, tstep = 0, iostep = Nsteps-1;

  int version = 3;
  int nsubs = (version==3) ? NSUBSTEPS:1;
  // time step
  for(tstep=0;tstep<(Nsteps+nsubs-1)/nsubs;++tstep){

    if(!((nsubs*tstep)%iostep)){ 
      hipEventRecord(tic);
    }
    
    // perform two updates
    if(version==0){
      dim3 T(TX,TY,1);
      dim3 B( (N+1+TX-1)/TX, (M+1+TY-1)/TY, 1);
      
      lbmUpdateV0 <<< B, T >>> (N, M, c, c_tau, c_nodeType, c_f, c_fnew);
      lbmUpdateV0 <<< B, T >>> (N, M, c, c_tau, c_nodeType, c_fnew, c_f);
    }

    if(version==2){
      dim3 T(TX,TY,1);
      dim3 B( (N+1+2*NHALO + (TX-2*NHALO)-1)/(TX-2*NHALO), (M+1+2*NHALO + (TY-2*NHALO)-1)/(TY-2*NHALO), 1);
      
      lbmUpdateV2 <<< B, T >>> (N, M, c, c_tau, c_nodeType, c_f, c_fnew);
      lbmUpdateV2 <<< B, T >>> (N, M, c, c_tau, c_nodeType, c_fnew, c_f);  
    }

    if(version==3){
      dim3 T(TX,TY,1);
      dim3 B( (N+1+2*NHALO + (TX-2*NHALO)-1)/(TX-2*NHALO), (M+1+2*NHALO + (TY-2*NHALO)-1)/(TY-2*NHALO), 1);
      
      lbmUpdateV3 <<< B, T >>> (N, M, c, c_tauInv, c_nodeType, c_f, c_fnew);
      lbmUpdateV3 <<< B, T >>> (N, M, c, c_tauInv, c_nodeType, c_fnew, c_f);
    }

    
    if(tstep>0 && !((nsubs*tstep)%iostep)){ // output an image every iostep
      hipEventRecord(toc);
      
      
      char fname[BUFSIZ];
      sprintf(fname, "bah%06d.png", (nsubs*tstep)/iostep);

      hipMemcpy(h_f, c_f, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyDeviceToHost);

      lbmOutput(fname, nodeType, rgb, alpha, c, dx, N, M, h_f);

      lbmCheck(N,M,h_f);

      float elapsed = 0;
      hipEventElapsedTime(&elapsed, tic, toc);
      elapsed /= 1000.f;

      double gnups = N*M*(2*nsubs/elapsed)/1.e9;
      printf("%02d, %02d, %02d, %05d, %5.4f; %%%% TX, TY, NSUBSTEPS, tstep, GNODES/s\n", TX, TY, NSUBSTEPS, tstep*nsubs, gnups);
    }
  }

  // output final result as image
  hipMemcpy(h_f, c_f, (N+2)*(M+2)*NSPECIES*sizeof(dfloat), hipMemcpyDeviceToHost);
  lbmOutput("bahFinal.png", nodeType, rgb, alpha, c, dx, N, M, h_f);

  exit(0);
  return 0;
}
  
